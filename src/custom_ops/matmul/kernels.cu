#include "hip/hip_runtime.h"
#include <vector>
#include <numeric>
#include "ops.hpp"

__device__ int getBatchOffset(const int* __restrict__ stridesOut,
                              const int* __restrict__ stridesIn, 
                              const int* __restrict__ shapeIn,
                              int linearIndexOut, int rank) {
    int batchOffset = 0;
    int dimIndex = 0;
    for (int i = 0; i < (rank - 2); i++) {
        dimIndex = linearIndexOut / stridesOut[i];
        linearIndexOut %= stridesOut[i];
        batchOffset += (shapeIn[i] == 1 ? 0 : dimIndex) * stridesIn[i];
    }
    return batchOffset;
}




template<typename scalar_t>
__global__ void cmatmul_forward_kernel(const scalar_t* __restrict__ inputA,
    const scalar_t* __restrict__ inputB, scalar_t* __restrict__ output,
    const int* __restrict__ stridesA, const int* __restrict__ stridesB,
    const int* __restrict__ stridesOut, const int* __restrict__ shapeA,
    const int* __restrict__ shapeB, int64_t rank) {

        int batch = blockIdx.z;
        int rowIndex = blockIdx.y * blockDim.y + threadIdx.y; 
        int colIndex = blockIdx.x * blockDim.x + threadIdx.x;

        int rowsA = shapeA[rank - 2];
        int colsA = shapeA[rank - 1];

        int rowsB = shapeB[rank - 2];
        int colsB = shapeB[rank - 1];

        if (rowIndex < rowsA && colIndex < colsB) {
            auto indexOut = batch * rowsA * colsB + rowIndex * colsB + colIndex;

            auto batchOffsetA = getBatchOffset(stridesOut, stridesA, shapeA, indexOut, rank);
            auto batchOffsetB = getBatchOffset(stridesOut, stridesB, shapeB, indexOut, rank);

            auto outValue = scalar_t(0);
            for (int i = 0; i < colsA; i++) {
                auto indexA = batchOffsetA * rowsA * colsA + rowIndex * colsA + i;
                auto indexB = batchOffsetB * rowsB * colsB + i * colsB + colIndex;

                outValue += inputA[indexA] * inputB[indexB];
            }
            output[indexOut] = outValue;
        }
}

at::Tensor torchlab::ops::matmul::unsqueezeToDim(const at::Tensor& t, int targetDim) {
    TORCH_CHECK(t.dim() <= targetDim,
    "expected input tensor to have at most ", targetDim, " dims, got ", t.dim());

    if (t.dim() == targetDim) {
        return t;
    }

    auto out = t;
    auto thisDim = t.dim();
    for (int i=0; i< targetDim - thisDim; i++) {
        out = out.unsqueeze(0);
    }
    return out;
}

void torchlab::ops::matmul::validateOpInput(const at::Tensor& inputA, const at::Tensor& inputB) {
    TORCH_CHECK(inputA.is_cuda(), "cmatmul::forward inputA must be a CUDA Tensor.");
    TORCH_CHECK(inputB.is_cuda(), "cmatmul::forward inputB must be a CUDA Tensor.");
    TORCH_CHECK(inputA.device() == inputB.device(),
            "cmatmul::forward both inputs must be on the same device");

    TORCH_CHECK(inputA.layout() == at::kStrided, "cmatmul::forward inputA must be a strided Tensor.");
    TORCH_CHECK(inputB.layout() == at::kStrided, "cmatmul::forward inputB must be a strided Tensor.");

    TORCH_CHECK(inputA.scalar_type() == at::kFloat || inputA.scalar_type() == at::kDouble,
                    "cmatmul::forward inputA must have float or double as a scalar type.");
    TORCH_CHECK(inputB.scalar_type() == at::kFloat || inputB.scalar_type() == at::kDouble,
                    "cmatmul::forward inputB must have float or double as a scalar type.");

    TORCH_CHECK(inputA.scalar_type() == inputB.scalar_type(),
                    "cmatmul::forward both inputs must have same scalar type.");

    TORCH_CHECK(inputA.numel() > 0, "cmatmul::forward inputA must have atleast 1 elements.");
    TORCH_CHECK(inputB.numel() > 0, "cmatmul::forward inputB must have atleast 1 elements.");

    auto shapeA = inputA.sizes();
    auto shapeB = inputB.sizes();

    TORCH_CHECK(shapeA.size() >= 2 && shapeB.size() >= 2 && shapeA[shapeA.size() - 1] == shapeB[shapeB.size() - 2],
                    "cmatmul::forward both inputs should be compatible for matrix multiplication.");
}

std::vector<int64_t> torchlab::ops::matmul::getOutputShape(const at::Tensor& matA, const at::Tensor& matB) {
    TORCH_CHECK(matA.dim() == matB.dim(),
                "cmatmul::forward tensors must have same number of dims after alignment, got ",
                matA.dim(), " vs ", matB.dim());

    auto matAShape = matA.sizes();
    auto matBShape = matB.sizes();

    auto outputShape = std::vector<int64_t>(matA.dim(), 1);
    outputShape[outputShape.size() - 1] = matBShape[matBShape.size() - 1];
    outputShape[outputShape.size() - 2] = matAShape[matAShape.size() - 2];

    for (int i = 0; i < outputShape.size() - 2; i++) {
        auto dimA = matAShape[i];
        auto dimB = matBShape[i];
        TORCH_CHECK(dimA == dimB || dimA == 1 || dimB == 1,
                    "cmatmul::forward the input tensors can't be broadcastable");
        outputShape[i] = std::max(dimA, dimB);
    }
    return outputShape;
}

at::Tensor torchlab::ops::matmul::forward(const at::Tensor& inputA, const at::Tensor& inputB) {
    validateOpInput(inputA, inputB);
    
    c10::cuda::CUDAGuard guard(inputA.device());
    auto stream = at::cuda::getCurrentCUDAStream();

    auto targetDim = std::max(inputA.dim(), inputB.dim());
    auto matA = unsqueezeToDim(inputA, targetDim);
    auto matB = unsqueezeToDim(inputB, targetDim);

    auto outputShape = getOutputShape(matA, matB);

    auto output = at::empty(outputShape, matA.options());

    auto stridesA = at::tensor(matA.strides(), matA.options().dtype(at::kLong));
    auto stridesB = at::tensor(matB.strides(), matB.options().dtype(at::kLong));
    auto stridesOut = at::tensor(output.strides(), output.options().dtype(at::kLong));

    int numThreadsRows = 32;
    int numThreadsCols = 32;
    //kernel call
    AT_DISPATCH_FLOATING_TYPES(matA.scalar_type(), "cmatmul::forward", [&]{
        const scalar_t* dataMatA = matA.data_ptr<scalar_t>();
        const scalar_t* dataMatB = matB.data_ptr<scalar_t>();

        scalar_t* dataOut = output.data_ptr<scalar_t>();

        int64_t batchCount = std::accumulate(outputShape.begin(), outputShape.end() - 2, int64_t(1), std::multiplies<int64_t>());

        dim3 gridConfig = dim3((outputShape[outputShape.size() - 2] + numThreadsRows - 1) / numThreadsRows,
                                (outputShape[outputShape.size() - 1] + numThreadsCols - 1) / numThreadsCols,
                                batchCount);
        dim3 blockConfig = dim3(numThreadsRows, numThreadsCols, 1);
        
        //kernel_launch_here
        //cmatmul_forward_kernel<scalar_t><<<gridConfig, blockConfig, 0, stream>>>(dataMatA, dataMatB, dataOut,
        //    matA.sizes()[matA.dim() - 2], matA.sizes()[matA.dim() - 1], matB.sizes()[matB.dim() - 1], batchCount, stridesOut, stridesA, stridesB, output.dim());

        C10_CUDA_KERNEL_LAUNCH_CHECK();
    });
    return output;
}